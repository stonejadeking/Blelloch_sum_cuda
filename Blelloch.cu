/*
A small code to test the blelloch and Hillis algrithm
Blelloch can only perform on size = 2^d array; thus a artitary array with size N should seperate into 2 arrary ->
one is 2^d(cloest to N for maximum efficiency) with Blelloch sum and another is N-2^n by using Hillis 
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include <cstdlib> 
#include <cmath> 
using namespace std;
int threads = 512;
int N=1536*17;
void exclusive_cpu(double* reference,double* host)
{
	for(int i=0;i<N;i++)
	{
		for(int ii=i-1;ii>=0;ii--)
		{
			reference[i]+=host[ii];
		}
	}
}

void generate_rand(double* h)
{
	for(int i = 0;i<N;i++){
		srand (i);
		h[i]=(double(rand())/RAND_MAX-0.5)*1000;
	}
	
}
__global__ void Belloch_sum_up(double* x, int i,int Nb)
{
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	int offset = 1<<i;
	if(idx>=Nb)return;
	if(idx%offset==offset-1&&idx>=offset/2)
	{
		x[idx]+=x[idx-(offset/2)];
	}
}
__global__ void Belloch_sum_down(double* x, int i,int Nb,int d)
{
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	if(idx>=Nb)return;
	int offset = 1<<i;
	if(idx%offset==offset-1&&idx>=offset/2)//idx%(offset)==0
	{
		double temp=x[idx];
		x[idx]+=x[idx-offset/2];
		x[idx-offset/2]=temp;
	}
}

__global__ void Hillis_sum(double* x,double* t, int i,int Nh)
{
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	if(idx>=Nh)return;
	if(idx>=i)t[idx]=x[idx]+x[idx-i];
	else t[idx]=x[idx];
	
}
__global__ void shift_offset (double*a,double b,int Nh)
{
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	if(idx>=Nh)return;
	a[idx]+=b;
}
int main (void)
{
	hipDeviceReset();
	hipSetDevice(0);
	int d = int(log2(double(N)));//d
	int Nb = 1<<d;//blelloch size, 2^d
	int bs = (Nb+threads-1)/threads;
	//cout<<"Nb="<<Nb<<endl;
	double host[N];
	double reference[N];
	double result_h[N];
	double *Belloch;
	generate_rand(host);
	memset(reference,0,sizeof(double)*N); 
	exclusive_cpu(reference,host);
	
	/*Blelloch part*/	
	hipMalloc((void**)&Belloch,sizeof(double)*Nb); 
	hipMemcpy(Belloch,host,sizeof(double)*Nb,hipMemcpyHostToDevice);
	for(int i=1;i<=d;i++){Belloch_sum_up<<<bs,threads>>>(Belloch,i,Nb);}
	hipMemset(&Belloch[Nb-1],0,sizeof(double));
	for(int i=d;i>=1;i--){Belloch_sum_down<<<bs,threads>>>(Belloch,i,Nb,d);}
	/**/
	
	/*Hillis part*/	
	if(Nb!=N)
	{
		int Nh = N-Nb;//Hillis size
		double offset=0;//offset of Hillis = sum of belloch part;
		hipMemcpy(&offset,&Belloch[Nb-1],sizeof(double),hipMemcpyDeviceToHost);
		double *Hillis,*temp;
		hipMalloc((void**)&Hillis,sizeof(double)*Nh); 
		hipMalloc((void**)&temp,sizeof(double)*Nh); 
		hipMemcpy(Hillis,&host[Nb-1],sizeof(double)*Nh,hipMemcpyHostToDevice);
		for(int i=1;i<Nh;i*=2)
		{
			Hillis_sum<<<bs,threads>>>(Hillis,temp,i,Nh);
			hipMemcpy(Hillis,temp,sizeof(double)*Nh,hipMemcpyDeviceToDevice);
		}
		shift_offset<<<bs,threads>>>(Hillis,offset,Nh);	
		hipMemcpy(&result_h[Nb],Hillis,sizeof(double)*Nh,hipMemcpyDeviceToHost);
		hipFree(Hillis);
		hipFree(temp);
	}
	hipMemcpy(result_h,Belloch,sizeof(double)*Nb,hipMemcpyDeviceToHost);
	double diff = 0;
	
	for(int i=0;i<N;i++)diff+=(result_h[i]-reference[i])*(result_h[i]-reference[i]);
	cout<<"N\t="<<N<<"\tstd divation\t=\t"<<sqrt(diff)/N<<endl;
	hipFree(Belloch);
	
}
